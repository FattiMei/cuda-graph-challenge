#include "hip/hip_runtime.h"
#include "reachability.hpp"


void cpuKernel(
	 int *nodePtrs
	,int *nodeNeighbors
	,int *nodeVisited
	,int *currLevelNodes
	,int *nextLevelNodes
	,const unsigned int numCurrLevelNodes
	,int *numNextLevelNodes){


	for(int i = 0; i < numCurrLevelNodes; ++i){
		const int u = currLevelNodes[i];
		const int neighborCount = nodePtrs[u+1] - nodePtrs[u];


		for(int j = 0; j < neighborCount; ++j){
			const int v = nodeNeighbors[nodePtrs[u] + j];

			if(nodeVisited[v] == 0){
				nodeVisited[v] = 1;


				// aggiunta alla coda
				nextLevelNodes[*numNextLevelNodes] = v;
				++(numNextLevelNodes);


				// è garantito che l'elemento non sia duplicato nella coda perchè è segnato come visitato
			}
		}
	}
}


void cpuReachability(CudaGraph G, std::vector<int> &nodeVisited){
}


__global__ void gpuKernel(
	 int *nodePtrs
	,int *nodeNeighbors
	,int *nodeVisited
	,int *currLevelNodes
	,int *nextLevelNodes
	,const unsigned int numCurrLevelNodes
	,int *numNextLevelNodes){


	// solo blocchi lineari
	int i = threadIdx.x + blockIdx.x * blockDim.x;


	if(i < numCurrLevelNodes){
		const int u = currLevelNodes[i];
		const int neighborCount = nodePtrs[u+1] - nodePtrs[u];


		for(int j = 0; i < neighborCount; ++j){
			const int v = nodeNeighbors[nodePtrs[u] + j];


			if(atomicCAS(nodeVisited + v, 0, 1) == 0){
				// sono il primo che fa la modifica, lo metto in coda
				int queuePtr = atomicAdd(numNextLevelNodes, 1);


				// sono l'unico che scrive qua dentro
				nextLevelNodes[queuePtr] = v;
			}
		}
	}
}
