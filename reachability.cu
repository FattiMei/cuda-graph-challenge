#include "hip/hip_runtime.h"
#include "reachability.hpp"
#include "error.hpp"


void cpuKernel(
	 int *nodePtrs
	,int *nodeNeighbors
	,int *nodeVisited
	,int *currLevelNodes
	,int *nextLevelNodes
	,const unsigned int numCurrLevelNodes
	,int *numNextLevelNodes){


	for(int i = 0; i < numCurrLevelNodes; ++i){
		const int u = currLevelNodes[i];
		const int neighborCount = nodePtrs[u+1] - nodePtrs[u];


		for(int j = 0; j < neighborCount; ++j){
			const int v = nodeNeighbors[nodePtrs[u] + j];

			if(nodeVisited[v] == 0){
				nodeVisited[v] = 1;


				// aggiunta alla coda
				nextLevelNodes[*numNextLevelNodes] = v;
				++(*numNextLevelNodes);


				// è garantito che l'elemento non sia duplicato nella coda perchè è segnato come visitato
			}
		}
	}
}


std::vector<int> cpuReachability(CSRGraph &G){
	std::vector<int> result(G.nodeCount, 0);


	int *currLevelNodes = new int[G.nodeCount];
	int *nextLevelNodes = new int[G.nodeCount];
	int *nodeVisited    = result.data();
	int numCurrLevelNodes;
	int numNextLevelNodes;


	// inizializzazione della coda
	numCurrLevelNodes = 1;
	currLevelNodes[0] = 0;


	while(numCurrLevelNodes != 0){
		numNextLevelNodes = 0;

		cpuKernel(
				G.nodePtrs
				,G.nodeNeighbors
				,nodeVisited
				,currLevelNodes
				,nextLevelNodes
				,numCurrLevelNodes
				,&numNextLevelNodes
				);

		numCurrLevelNodes = numNextLevelNodes;
		std::swap(currLevelNodes, nextLevelNodes);
	}


	delete[] currLevelNodes;
	delete[] nextLevelNodes;


	return result;
}


__global__ void gpuKernel(
	 int *nodePtrs
	,int *nodeNeighbors
	,int *nodeVisited
	,int *currLevelNodes
	,int *nextLevelNodes
	,const unsigned int numCurrLevelNodes
	,int *numNextLevelNodes){


	// solo blocchi lineari
	int i = threadIdx.x + blockIdx.x * blockDim.x;


	if(i < numCurrLevelNodes){
		const int u = currLevelNodes[i];
		const int neighborCount = nodePtrs[u+1] - nodePtrs[u];


		for(int j = 0; i < neighborCount; ++j){
			const int v = nodeNeighbors[nodePtrs[u] + j];


			// riga maledetta, mi ha tenuto 2 ore di debugging
			//if(atomicCAS(nodeVisited + v, 0, 1) == 0){
			if(atomicCAS(&nodeVisited[v], 0, 1) == 0){
				// sono il primo che fa la modifica, lo metto in coda
				int queuePtr = atomicAdd(numNextLevelNodes, 1);


				// sono l'unico che scrive qua dentro
				nextLevelNodes[queuePtr] = v;
			}
		}
	}
}


__global__ void addKernel(
	 int *nodePtrs
	,int *nodeNeighbors
	,int *nodeVisited
	,int *currLevelNodes
	,int *nextLevelNodes
	,const unsigned int numCurrLevelNodes
	,int *numNextLevelNodes){

  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if(i < numCurrLevelNodes){
    const int u = currLevelNodes[i];
    const int neighborCount = nodePtrs[u+1] - nodePtrs[u];

      for(int j = 0; j < neighborCount; ++j){
        const int v = nodeNeighbors[nodePtrs[u] + j];

        if(atomicCAS(&nodeVisited[v], 0, 1) == 0){

          int queuePtr = atomicAdd(numNextLevelNodes, 1);
          nextLevelNodes[queuePtr] = v;

        }
      }
	}
}


std::vector<int> gpuReachability(CSRGraph &G){
	std::vector<int> result(G.nodeCount, 0);


	/*
		Nome				Tipo				Relazione
		nodePtrs			int[nodeCount+1]	HOST -> DEVICE
		nodeNeighbors		int[nodeCount]		HOST -> DEVICE
		nodeVisited			int[nodeCount]		DEVICE -> HOST (ma ha bisogno di essere inizializzato)
		currLevelNodes		int[nodeCount]		DEVICE
		nextLevelNodes		int[nodeCount]		DEVICE (ma ha bisogno di essere inizializzato)
		numNextLevelNodes	int*				DEVICE (ma ha bisogno di essere inizializzato)
		numCurrLevelNodes	int					HOST -> parametro della funzione al device
	*/


	int *nodePtrs = NULL;
	int *nodeNeighbors = NULL;
	int *nodeVisited = NULL;
	int *currLevelNodes = NULL;
	int *nextLevelNodes = NULL;
	int *numNextLevelNodes = NULL;
	int numCurrLevelNodes;


	CHECK_CUDA_ERROR(hipMalloc(&nodePtrs,       sizeof(int) * (G.nodeCount+1)));
	CHECK_CUDA_ERROR(hipMalloc(&nodeNeighbors,  sizeof(int) * G.edgeCount));
	CHECK_CUDA_ERROR(hipMalloc(&nodeVisited,    sizeof(int) * G.nodeCount));
	CHECK_CUDA_ERROR(hipMalloc(&currLevelNodes, sizeof(int) * G.nodeCount));
	CHECK_CUDA_ERROR(hipMalloc(&nextLevelNodes, sizeof(int) * G.nodeCount));
	CHECK_CUDA_ERROR(hipMalloc(&numNextLevelNodes, sizeof(int)));


	CHECK_CUDA_ERROR(hipMemcpy(nodePtrs,           G.nodePtrs, sizeof(int) * (G.nodeCount+1), hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(nodeNeighbors, G.nodeNeighbors, sizeof(int) * G.edgeCount,     hipMemcpyHostToDevice));


	// inizializzazione della coda
	numCurrLevelNodes = 1;


	// currLevelNodes[0] = 0;
	CHECK_CUDA_ERROR(hipMemset(currLevelNodes, 0, sizeof(int)));
	CHECK_CUDA_ERROR(hipMemset(nodeVisited, 0, sizeof(int) * G.nodeCount));


	int iter = 0;
	while(numCurrLevelNodes != 0){
		// numNextLevelNodes = 0;
		CHECK_CUDA_ERROR(hipMemset(numNextLevelNodes, 0, sizeof(int)));

    int threadsPerBlock = 1024;
    int blockSize = (numCurrLevelNodes + threadsPerBlock - 1) / threadsPerBlock;

		addKernel<<<blockSize, threadsPerBlock>>>(
				 nodePtrs
				,nodeNeighbors
				,nodeVisited
				,currLevelNodes
				,nextLevelNodes
				,numCurrLevelNodes
				,numNextLevelNodes
				);
		CHECK_CUDA_ERROR(hipPeekAtLastError());

		// numCurrLevelNodes = *numNextLevelNodes;
		CHECK_CUDA_ERROR(hipMemcpy(&numCurrLevelNodes, numNextLevelNodes, sizeof(int), hipMemcpyDeviceToHost));

		std::swap(currLevelNodes, nextLevelNodes);

    //std::cout << numCurrLevelNodes << std::endl;
		++iter;
	}

  CHECK_CUDA_ERROR(hipMemcpy(result.data(), nodeVisited, G.nodeCount * sizeof(int), hipMemcpyDeviceToHost));


	CHECK_CUDA_ERROR(hipFree(nodePtrs));
	CHECK_CUDA_ERROR(hipFree(nodeNeighbors));
	CHECK_CUDA_ERROR(hipFree(nodeVisited));
	CHECK_CUDA_ERROR(hipFree(currLevelNodes));
	CHECK_CUDA_ERROR(hipFree(nextLevelNodes));
	CHECK_CUDA_ERROR(hipFree(numNextLevelNodes));


	return result;
}
