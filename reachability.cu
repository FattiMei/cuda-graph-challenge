#include "hip/hip_runtime.h"
#include "reachability.hpp"


void cpuKernel(
	 int *nodePtrs
	,int *nodeNeighbors
	,int *nodeVisited
	,int *currLevelNodes
	,int *nextLevelNodes
	,const unsigned int numCurrLevelNodes
	,int *numNextLevelNodes){


	for(int i = 0; i < numCurrLevelNodes; ++i){
		const int u = currLevelNodes[i];
		const int neighborCount = nodePtrs[u+1] - nodePtrs[u];


		for(int j = 0; j < neighborCount; ++j){
			const int v = nodeNeighbors[nodePtrs[u] + j];

			if(nodeVisited[v] == 0){
				nodeVisited[v] = 1;


				// aggiunta alla coda
				nextLevelNodes[*numNextLevelNodes] = v;
				++(*numNextLevelNodes);


				// è garantito che l'elemento non sia duplicato nella coda perchè è segnato come visitato
			}
		}
	}
}


std::vector<int> cpuReachability(CudaGraph &G){
	std::vector<int> result(G.nodeCount, 0);


	int *currLevelNodes = new int[G.nodeCount];
	int *nextLevelNodes = new int[G.nodeCount];
	int *nodeVisited    = result.data();
	int numCurrLevelNodes;
	int numNextLevelNodes;


	// inizializzazione della coda
	numCurrLevelNodes = 1;
	currLevelNodes[0] = 0;


	while(numCurrLevelNodes != 0){
		numNextLevelNodes = 0;

		cpuKernel(
				G.nodePtrs
				,G.nodeNeighbors
				,nodeVisited
				,currLevelNodes
				,nextLevelNodes
				,numCurrLevelNodes
				,&numNextLevelNodes
				);

		numCurrLevelNodes = numNextLevelNodes;
		std::swap(currLevelNodes, nextLevelNodes);
	}


	delete[] currLevelNodes;
	delete[] nextLevelNodes;


	return result;
}


__global__ void gpuKernel(
	 int *nodePtrs
	,int *nodeNeighbors
	,int *nodeVisited
	,int *currLevelNodes
	,int *nextLevelNodes
	,const unsigned int numCurrLevelNodes
	,int *numNextLevelNodes){


	// solo blocchi lineari
	int i = threadIdx.x + blockIdx.x * blockDim.x;


	if(i < numCurrLevelNodes){
		const int u = currLevelNodes[i];
		const int neighborCount = nodePtrs[u+1] - nodePtrs[u];


		for(int j = 0; i < neighborCount; ++j){
			const int v = nodeNeighbors[nodePtrs[u] + j];


			if(atomicCAS(nodeVisited + v, 0, 1) == 0){
				// sono il primo che fa la modifica, lo metto in coda
				int queuePtr = atomicAdd(numNextLevelNodes, 1);


				// sono l'unico che scrive qua dentro
				nextLevelNodes[queuePtr] = v;
			}
		}
	}
}


std::vector<int> gpuReachability(CudaGraph &G){
	std::vector<int> result(G.nodeCount, 0);


	/*
		Nome				Tipo			Relazione
		nodePtrs			int[nodeCount]	HOST -> DEVICE
		nodeNeighbors		int[nodeCount]	HOST -> DEVICE
		nodeVisited			int[nodeCount]	DEVICE -> HOST (ma ha bisogno di essere inizializzato)
		currLevelNodes		int[nodeCount]	DEVICE
		nextLevelNodes		int[nodeCount]	DEVICE (ma ha bisogno di essere inizializzato)
		numNextLevelNodes	int*			DEVICE (ma ha bisogno di essere inizializzato)
		numCurrLevelNodes	int				HOST -> parametro della funzione al device
	*/


	int *nodePtrs = NULL;
	int *nodeNeighbors = NULL;
	int *nodeVisited = NULL;
	int *currLevelNodes = NULL;
	int *nextLevelNodes = NULL;
	int *numNextLevelNodes = NULL;
	int numCurrLevelNodes;


	hipMalloc(&nodePtrs,       sizeof(int) * G.nodeCount);
	hipMalloc(&nodeNeighbors,  sizeof(int) * G.nodeCount);
	hipMalloc(&nodeVisited,    sizeof(int) * G.nodeCount);
	hipMalloc(&currLevelNodes, sizeof(int) * G.nodeCount);
	hipMalloc(&nextLevelNodes, sizeof(int) * G.nodeCount);
	hipMalloc(&numNextLevelNodes, sizeof(int));


	hipMemcpy(nodePtrs,           G.nodePtrs, sizeof(int) * G.nodeCount, hipMemcpyHostToDevice);
	hipMemcpy(nodeNeighbors, G.nodeNeighbors, sizeof(int) * G.nodeCount, hipMemcpyHostToDevice);


	// inizializzazione della coda
	numCurrLevelNodes = 1;

	// currLevelNodes[0] = 0;
	hipMemset(currLevelNodes, 0, sizeof(int));


	while(numCurrLevelNodes != 0){
		// numNextLevelNodes = 0;
		hipMemset(numNextLevelNodes, 0, sizeof(int));

		gpuKernel<<<1, 1024>>>(
				 nodePtrs
				,nodeNeighbors
				,nodeVisited
				,currLevelNodes
				,nextLevelNodes
				,numCurrLevelNodes
				,numNextLevelNodes
				);

		// numCurrLevelNodes = *numNextLevelNodes;
		hipMemcpy(&numCurrLevelNodes, numNextLevelNodes, sizeof(int), hipMemcpyDeviceToHost);

		std::swap(currLevelNodes, nextLevelNodes);
	}


	hipFree(nodePtrs);
	hipFree(nodeNeighbors);
	hipFree(nodeVisited);
	hipFree(currLevelNodes);
	hipFree(nextLevelNodes);
	hipFree(numNextLevelNodes);


	return result;
}
